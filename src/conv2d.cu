#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <cstdlib>
#include <iostream>

// CUDA kernel for 2D convolution
__global__ void conv2dKernel(unsigned char* dst, const unsigned char* img, const int* kernel, size_t krows, size_t kcols, size_t irows, size_t icols, size_t channels) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < irows && col < icols) {
        int dx = kcols / 2;
        int dy = krows / 2;
        for (size_t c = 0; c < channels; c++) {
            float tmp = 0.0f;
            for (int k = 0; k < krows; ++k) {
                for (int l = 0; l < kcols; ++l) {
                    int x = col - dx + l;
                    int y = row - dy + k;
                    if (x >= 0 && x < icols && y >= 0 && y < irows) {
                        tmp += img[(y * icols + x) * channels + c] * kernel[k * kcols + l];
                    }
                }
            }
            dst[(row * icols + col) * channels + c] = fminf(fmaxf(tmp, 0.0f), 255.0f);
        }
    }
}

// Function to perform 2D convolution using CUDA
unsigned char* conv2dWithCuda(unsigned char* img, const int* kernel, size_t krows, size_t kcols, size_t irows, size_t icols, size_t channels) {
    unsigned char* dev_img = nullptr;
    int* dev_kernel = nullptr;
    unsigned char* dev_dst = nullptr;
    unsigned char* result = nullptr;
    hipError_t cudaStatus;
    size_t imgSize = irows * icols * channels * sizeof(unsigned char);
    size_t kernelSize = krows * kcols * sizeof(int);

    // Allocate host memory for the result
    result = (unsigned char*)std::malloc(imgSize);
    if (!result) {
        std::cerr << "Host memory allocation failed!" << std::endl;
        goto Error;
    }

    // Choose which GPU to run on
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipSetDevice failed! Do you have a CUDA-capable GPU installed?" << std::endl;
        goto Error;
    }

    // Allocate GPU buffers for the image, kernel, and output
    cudaStatus = hipMalloc((void**)&dev_img, imgSize);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for dev_img!" << std::endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_kernel, kernelSize);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for dev_kernel!" << std::endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_dst, imgSize);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for dev_dst!" << std::endl;
        goto Error;
    }

    // Copy image and kernel from host to device
    cudaStatus = hipMemcpy(dev_img, img, imgSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed for dev_img!" << std::endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_kernel, kernel, kernelSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed for dev_kernel!" << std::endl;
        goto Error;
    }

    // Launch the convolution kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((icols + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (irows + threadsPerBlock.y - 1) / threadsPerBlock.y);
    conv2dKernel << <numBlocks, threadsPerBlock >> > (dev_dst, dev_img, dev_kernel, krows, kcols, irows, icols, channels);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "conv2dKernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching conv2dKernel!" << std::endl;
        goto Error;
    }

    // Copy the result from device to host
    cudaStatus = hipMemcpy(result, dev_dst, imgSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed for result!" << std::endl;
        goto Error;
    }

Error:
    hipFree(dev_img);
    hipFree(dev_kernel);
    hipFree(dev_dst);

    if (cudaStatus != hipSuccess) {
        std::free(result);
        result = nullptr;
    }

    return result;
}
